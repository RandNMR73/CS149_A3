#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void copy(int N, int* input, int* result) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < N) {
        result[index] = input[index];
    }
}

__global__ void scan_upsweep_kernel(int num_iter, int two_d, int two_dplus1, int* result) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_iter) {
        int i = index * two_dplus1;
        result[i+two_dplus1-1] += result[i+two_d-1];
    }

    __syncthreads();

    if (num_iter == 1) {
		result[2 * two_d - 1] = 0;
	}
}

__global__ void scan_downsweep_kernel(int num_iter, int two_d, int two_dplus1, int* result) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < num_iter) {
        int i = index * two_dplus1;
        int t = result[i+two_d-1];
        result[i+two_d-1] = result[i+two_dplus1-1];
        result[i+two_dplus1-1] += t;
    }
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result) {

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    // copy input to result
    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    copy<<<blocks, THREADS_PER_BLOCK>>>(N, input, result);
	
    // upsweep phase
    for (int two_d = 1; two_d <= N/2; two_d*=2) {
        int two_dplus1 = 2*two_d;
        int num_iter = N / two_dplus1;
        blocks = (num_iter + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
		
        scan_upsweep_kernel<<<blocks, THREADS_PER_BLOCK>>>(num_iter, two_d, two_dplus1, result);
    }

    // downsweep phase
    for (int two_d = N/2; two_d >= 1; two_d /= 2) {
        int two_dplus1 = 2*two_d;
        int num_iter = N / two_dplus1;
        blocks = (num_iter + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        scan_downsweep_kernel<<<blocks, THREADS_PER_BLOCK>>>(num_iter, two_d, two_dplus1, result);
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray) {

    int* device_result;
    int* device_input;
    int N = end - inarray;

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


__global__ void pair_equal_adjacent(int N, int* input, int* output) {
    
    __shared__ int input_copy[THREADS_PER_BLOCK + 1];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int copy_index = threadIdx.x;

    if (index < N - 1) {
        if (copy_index < THREADS_PER_BLOCK - 1) {
            input_copy[copy_index] = input[index];
        } else {
            input_copy[copy_index] = input[index];
            input_copy[copy_index + 1] = input[index + 1];
        }
    } else if (index == N - 1) {
    	input_copy[copy_index] = input[index];
    }

    __syncthreads();

    if (index < N - 1) {
        output[index] = (input_copy[copy_index] == input_copy[copy_index + 1]);
    }
}

__global__ void update_pair_index(int N, int* input, int* output) {
    
    __shared__ int input_copy[THREADS_PER_BLOCK + 1];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int copy_index = threadIdx.x;
    
    if (index < N - 1) {
        if (copy_index < THREADS_PER_BLOCK - 1) {
            input_copy[copy_index] = input[index];
        } else {
            input_copy[copy_index] = input[index];
            input_copy[copy_index + 1] = input[index + 1];
        }
    } else if (index == N - 1) {
    	input_copy[copy_index] = input[index];
    }

    __syncthreads();

    if (index < N - 1) {
        if (input_copy[copy_index + 1] > input_copy[copy_index]) {
            output[input_copy[copy_index]] = index;
        }
    }
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    int blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    std::cout << "before pair_equal_adjacent\n";
    // equal pair phase
    pair_equal_adjacent<<<blocks, THREADS_PER_BLOCK>>>(length, device_input, device_output);
    
    int* input = new int[length];
    hipMemcpy(input, device_output, length * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < length; i++) std::cout << input[i] << " ";
    std::cout << "\n";

    // exclusive scan phase
    std::cout << "before exclusive scan\n";
    exclusive_scan(device_output, length, device_input);
    std::cout << "after exclusive scan\n";

    hipMemcpy(input, device_input, length * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < length; i++) std::cout << input[i] << " ";
    std::cout << "\n";
    
    // update output phase
    int num_repeats = 0;
    hipMemcpy(&num_repeats, device_input + length - 1, 1 * sizeof(int), hipMemcpyDeviceToHost);
    update_pair_index<<<blocks, THREADS_PER_BLOCK>>>(length, device_input, device_output);
    std::cout << "after update_pair_index\n";
    
    delete[] input;
    return num_repeats;
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < length; i++) std::cout << input[i] << " ";
    std::cout << "\n";

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}


void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
